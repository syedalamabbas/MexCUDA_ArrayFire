
#include <hip/hip_runtime.h>
/** @file processMandelbrotElement.cu
 *
 * Copyright 2010 The Mathworks, Inc.
 * $Revision: 1$
 * $Date: 2010-11-08$
 */

/** Work out which piece of the global array this thread should operate on */ 
__device__ size_t calculateGlobalIndex() {
    // Which block are we?
    size_t const globalBlockIndex = blockIdx.x + blockIdx.y * gridDim.x;
    // Which thread are we within the block?
    size_t const localThreadIdx = threadIdx.x + blockDim.x * threadIdx.y;
    // How big is each block?
    size_t const threadsPerBlock = blockDim.x*blockDim.y;
    // Which thread are we overall?
    return localThreadIdx + globalBlockIndex*threadsPerBlock;

}

/** The actual Mandelbrot algorithm for a single location */ 
__device__ double doIterations( double const realPart0, 
                                      double const imagPart0, 
                                      double const escapeRadius, 
                                      unsigned int const maxIters ) {
    // Initialise: z = z0
    double const escapeRadius2 = escapeRadius*escapeRadius;
    double realPart = realPart0;
    double imagPart = imagPart0;
    unsigned int count = 0;

    // Loop until escape
    while ( ( count <= maxIters )
            && ((realPart*realPart + imagPart*imagPart) <= escapeRadius2) ) {
        ++count;
        // Update: z = z*z + z0;
        double const oldRealPart = realPart;
        realPart = realPart*realPart - imagPart*imagPart + realPart0;
        imagPart = 2.0*oldRealPart*imagPart + imagPart0;
    }

    // Correct final position for smooth shading
    double const absZ2 = ( realPart*realPart + imagPart*imagPart );
    if (absZ2<escapeRadius2) {
        return double(count) + 1.0 - log( log( escapeRadius2 ) / 2.0 ) / log(2.0);
    } else {
        return double(count) + 1.0 - log( log( absZ2 ) / 2.0 ) / log(2.0);
    }
}


/** Main entry point.
 * Works out where the current thread should read/write to global memory
 * and calls doIterations to do the actual work.
 */
__global__ void processMandelbrotElement( 
                      double * out, 
                      const double * x, 
                      const double * y,
                      const double escapeRadius, 
                      const unsigned int maxIters, 
                      const unsigned int numel ) {
    // Work out which thread we are
    size_t const globalThreadIdx = calculateGlobalIndex();

    // If we're off the end, return now
    if (globalThreadIdx >= numel) {
        return;
    }
    
    // Get our X and Y coords
    double const realPart0 = x[globalThreadIdx];
    double const imagPart0 = y[globalThreadIdx];

    // Run the itearations on this location
    double const count = doIterations( realPart0, imagPart0, escapeRadius, maxIters );
    out[globalThreadIdx] = log( count );
}
