#include "hip/hip_runtime.h"
/*
* Example of how to use the mxGPUArray API in a MEX file.  This example shows
* how to write a MEX function that takes a gpuArray as input and returns a
* gpuArray output for Mandelbrot Example solution, e.g. B=mexFunction(A).
*
* by Syed Alam Abbas, 5/23/2015
*/
#include <arrayfire.h>
#include <af/util.h>
#include "hip/hip_runtime.h"
#include "mex.h"
#include "gpu/mxGPUArray.h"

using namespace af;
/*
* Host code
*/
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, mxArray const *prhs[])
{
	/* Initialize the MathWorks GPU API. */
	mxInitGPU();

	mexPrintf("Executing custom mex for computing Mandelbrot set using ArrayFire GPU accelerated library !\n");
	
	// Validate the input
	if ( nrhs < 5 || nlhs < 1 ) {
		mexErrMsgTxt("Expected 5 inputs and 1 output.");
	}

	/*Input Variables*/
	mxGPUArray const *xVal;
	mxGPUArray const *yVal;
	mxGPUArray const *Count;
	mxGPUArray*  intermediateTemp;
	int d_MaxIterations;
	int d_NumberOfElements;

	/*Device Temp Variables */
	const double* d_xVal;
	const double* d_yVal;
	const double* d_Count;
	
	/*Output Variable*/
	double* d_CountOutImage;
	
	/* Collect the input data from MATLAB MexArray RHS */
	xVal = mxGPUCreateFromMxArray(prhs[0]);
	yVal = mxGPUCreateFromMxArray(prhs[1]);   
	Count = mxGPUCreateFromMxArray(prhs[2]);
	d_MaxIterations = (size_t) mxGetScalar(prhs[3]);
	d_NumberOfElements = (size_t)mxGetScalar(prhs[4]);


	/* extract a pointer to the input data on the device.*/
	d_xVal = (double const *)(mxGPUGetDataReadOnly(xVal));
	d_yVal = (double const *)(mxGPUGetDataReadOnly(yVal));
	d_Count = (double const *)(mxGPUGetDataReadOnly(Count));

	/* Create a GPUArray to hold the result and get its underlying pointer. */
	intermediateTemp = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(Count),
		mxGPUGetDimensions(Count),
		mxGPUGetClassID(Count),
		mxGPUGetComplexity(Count),
		MX_GPU_DO_NOT_INITIALIZE);
	d_CountOutImage = (double *)(mxGPUGetData(intermediateTemp));

	/* Copy Input Value from Count */
	hipMemcpy(d_CountOutImage, d_Count, d_NumberOfElements* d_NumberOfElements* sizeof(double), hipMemcpyDeviceToDevice);

	/* Using ArrayFire Code for processing Now */
	array array_X(d_NumberOfElements, d_xVal);
	array array_Y(d_NumberOfElements, d_yVal);
	array CountImage(d_NumberOfElements, d_NumberOfElements, d_CountOutImage);

	array array_X_Tiled = tile(array_X, 1, d_NumberOfElements);
	array array_Y_Tiled = tile(array_Y.T(), d_NumberOfElements, 1);

	array Z_0_complex = complex(array_X_Tiled, array_Y_Tiled);
	array Solution(Z_0_complex);
	for (int ii = 0; ii < (d_MaxIterations + 1); ii++)
	{
		Solution = Solution * Solution + Z_0_complex;
		CountImage = CountImage + (abs(Solution) <= 2);
	}
	CountImage = log(CountImage);
	
	/* Copy Processed Values to Output*/
	double* d_Processed = CountImage.device<double>();
	hipMemcpy(d_CountOutImage, d_Processed, d_NumberOfElements* d_NumberOfElements* sizeof(double), hipMemcpyDeviceToDevice);

	/* Wrap the result up as a MATLAB gpuArray for return. */
	plhs[0] = mxGPUCreateMxArrayOnGPU(intermediateTemp);
	
	/*
	* The mxGPUArray pointers are host-side structures that refer to device
	* data. These must be destroyed before leaving the MEX function.
	*/
	mxGPUDestroyGPUArray(yVal);
	mxGPUDestroyGPUArray(xVal);
	mxGPUDestroyGPUArray(Count);

	mexPrintf("Finished processing custom CUDA mex with ArrayFire, Status = Success\n");
}
